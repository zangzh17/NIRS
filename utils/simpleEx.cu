
#include <hip/hip_runtime.h>
/*
* Add a constant to a vector.
*/
__global__ void addToVector(float * pi, float c, int vecLen)  {
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   if (idx < vecLen) {
       pi[idx] += c;
   }
}